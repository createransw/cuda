#include "hip/hip_runtime.h"
/* Jacobi-3 program */

#include <ctime>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <ostream>
#include <thrust/device_vector.h>

#define SAFE_CALL(err) do {                     \
    if (err != 0) {                             \
        std::cerr << "ERROR [" << __FILE__ << "] in line " << __LINE__ << ": " <<       hipGetErrorString(err) << std::endl;   \
        exit(1);                                \
    }                                           \
} while(0)


#define Max(a, b) ((a) > (b) ? (a) : (b))
#define fabs(a) ((a) > 0 ? (a) : -(a))

#define A(i, j, k) A[((i) * L + (j)) * L + (k)]
#define B(i, j, k) B[((i) * L + (j)) * L + (k)]
#define eps(i, j, k) eps[((i) * L + (j)) * L + (k)]


// #define L 885
#define L 700
#define ITMAX 100

double eps;
double MAXEPS = 0.5;

__global__ void function(const double *A, double *B, double *eps) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;

    if ((i > 0) && (i < L - 1)) {
        if ((j > 0) && (j < L - 1)) {
            if ((k > 0) && (k < L - 1)) {
                B(i, j, k) = (A(i - 1, j, k) + A(i, j - 1, k) + A(i, j, k - 1) + A(i, j, k + 1) + A(i, j + 1, k) + A(i + 1, j, k)) / 6.0;
                eps(i, j, k) = fabs(B(i, j, k) - A(i, j, k));
            }
        }
    }
}

__global__ void difference_ab(double *A, const double *B, double *eps) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;

    if ((i > 0) && (i < L - 1)) {
        if ((j > 0) && (j < L - 1)) {
            if ((k > 0) && (k < L - 1)) {
                eps(i, j, k) = fabs(B(i, j, k) - A(i, j, k));
                A(i, j, k) = B(i, j, k);
            }
        }
    }
}

double dev(const double *A, const double *B) {
    double delta = 0.0;
    for (int i = 1; i < L - 1; i++)
        for (int j = 1; j < L - 1; j++)
            for (int k = 1; k < L - 1; k++)
            {
                double tmp = fabs(B(i, j, k) - A(i, j, k));
                delta = Max(tmp, delta);
            }
    return delta;
}

void set(double *A, double *B) {
    for (int i = 0; i < L; i++)
        for (int j = 0; j < L; j++)
            for (int k = 0; k < L; k++)
            {
                A(i, j, k)= 0;
                if (i == 0 || j == 0 || k == 0 || i == L - 1 || j == L - 1 || k == L - 1)
                    B(i, j, k) = 0;
                else
                    B(i, j, k) = 4 + i + j + k;
            }    
}

int main(int an, char **as)
{
    std::cout << "usage:\t\tyacoby -[cg]" << std::endl;

    bool CPU = false;
    bool GPU = true;
    if (an >= 2) {
        GPU = false;
        for (int i = 0; as[1][i] != '\0'; ++i) {
            if (as[1][i] == 'g') GPU = true;
            if (as[1][i] == 'c') CPU = true;
        }
    }

    const long size = L * L * L * sizeof(double);
    
    double *A = (double*)malloc(size);
    double *B = (double*)malloc(size);

    float cpu_time = 0;
    if (CPU) {
        set(A, B);

        /* iteration loop */
        clock_t startt = clock();
        for (int it = 1; it <= ITMAX; it++) {
            eps = 0;
            
            for (int i = 1; i < L - 1; i++)
                for (int j = 1; j < L - 1; j++)
                    for (int k = 1; k < L - 1; k++)
                    {
                        double tmp = fabs(B(i, j, k) - A(i, j, k));
                        eps = Max(tmp, eps);
                        A(i, j, k) = B(i, j, k);
                    }

            for (int i = 1; i < L - 1; i++)
                for (int j = 1; j < L - 1; j++)
                    for (int k = 1; k < L - 1; k++)
                        B(i, j, k) = (A(i - 1, j, k) + A(i, j - 1, k) + A(i, j, k - 1) + A(i, j, k + 1) + A(i, j + 1, k) + A(i + 1, j, k)) / 6.0f;

            if (eps < MAXEPS)
                break;
        }
        clock_t endt = clock();

        cpu_time = float(endt - startt) / CLOCKS_PER_SEC;
    }

    double *A_host = (double*)malloc(size);
    double *B_host = (double*)malloc(size);

    float gpu_time = 0;
    if (GPU) {
        int deviceCount=0;
        hipGetDeviceCount( &deviceCount ); // число доступных устройств
        if (deviceCount < 1) exit(1);
        hipSetDevice(0); // Выбрать для работы заданное устройство
        
        set(A_host, B_host);

        double *A_device, *B_device;
        SAFE_CALL(hipMalloc((void**)&A_device, size));
        SAFE_CALL(hipMemcpy(A_device, A_host, size, hipMemcpyHostToDevice));
        SAFE_CALL(hipMalloc((void**)&B_device, size));
        SAFE_CALL(hipMemcpy(B_device, B_host, size, hipMemcpyHostToDevice));

        thrust::device_vector<double> diff(L * L * L);
        double *ptrdiff = thrust::raw_pointer_cast(&diff[0]);
        double eps = 0.0;


        dim3 blockDim = dim3(32, 4, 4);
        //int block = blockDim.x * blockDim.y * blockDim.z;
        dim3 gridDim = dim3(L / 32 + 1, L / 4 + 1, L / 4 + 1);

       
        bool flg = true;


        hipEvent_t startt, endt;
        hipEventCreate(&startt);
        hipEventCreate(&endt);

        hipEventRecord(startt, 0);

        difference_ab<<<gridDim, blockDim>>>(A_device, B_device, ptrdiff);
        flg = not flg;
        eps = thrust::reduce(diff.begin(), diff.end(), 0.0, thrust::maximum<double>());
        /* iteration loop */
        for (int it = 1; it <= ITMAX - 1; it++) {
            if (eps < MAXEPS)
                break;
            if (flg)
                function<<<gridDim, blockDim>>>(A_device, B_device, ptrdiff);
            else 
                function<<<gridDim, blockDim>>>(B_device, A_device, ptrdiff);
            flg = not flg;
            eps = thrust::reduce(diff.begin(), diff.end(), 0.0, thrust::maximum<double>());
        }
        hipEventRecord(endt, 0);

        hipEventSynchronize(endt);
        hipEventElapsedTime(&gpu_time, startt, endt);
        hipEventDestroy(startt);
        hipEventDestroy(endt);

        if (flg)
            SAFE_CALL(hipMemcpy(A_host, A_device, size, hipMemcpyDeviceToHost));
        else
            SAFE_CALL(hipMemcpy(A_host, B_device, size, hipMemcpyDeviceToHost));

        SAFE_CALL(hipFree(A_device));
        SAFE_CALL(hipFree(B_device));
    }


    if (CPU && GPU) {
        std::cout << "cpu time = " << cpu_time << std::endl;
        std::cout << "gpu time = " << gpu_time * 0.001 <<  std::endl;
        std::cout << "decrease = " << cpu_time / gpu_time * 1000<< std::endl;
        std::cout << "maksimum deviation = " << dev(A, A_host) << std::endl;
    } else if (CPU) {
        std::cout << "cpu time = " << cpu_time << std::endl;
    } else if (GPU) {
        std::cout << "gpu time = " << gpu_time * 0.001 << std::endl;
    }

    free(A);
    free(A_host);
    free(B);
    free(B_host);

    return 0;
}
