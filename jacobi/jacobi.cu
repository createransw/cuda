#include "hip/hip_runtime.h"
/* Jacobi-3 program */

#include <ctime>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <ostream>
#include <thrust/device_vector.h>

#define SAFE_CALL(err) do {                     \
    if (err != 0) {                             \
        std::cerr << "ERROR [" << __FILE__ << "] in line " << __LINE__ << ": " <<       hipGetErrorString(err) << std::endl;   \
        exit(1);                                \
    }                                           \
} while(0)


#define Max(a, b) ((a) > (b) ? (a) : (b))
#define fabs(a) ((a) > 0 ? (a) : -(a))

#define A(i, j, k) A[((i) * L + (j)) * L + (k)]
#define B(i, j, k) B[((i) * L + (j)) * L + (k)]
#define eps(i, j, k) eps[((i) * L + (j)) * L + (k)]


// #define L 885
#define L 200
#define ITMAX 100

double eps;
double MAXEPS = 0.5;

__global__ void function(const double *A, double *B) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;

    if ((i > 0) && (i < L - 1)) {
        if ((j > 0) && (j < L - 1)) {
            if ((k > 0) && (k < L - 1)) {
                B(i, j, k) = (A(i - 1, j, k) + A(i, j - 1, k) + A(i, j, k - 1) + A(i, j, k + 1) + A(i, j + 1, k) + A(i + 1, j, k)) / 6.0;
            }
        }
    }
}

double dev(const double *A, const double *B) {
    double delta = 0.0;
    for (int i = 1; i < L - 1; i++)
        for (int j = 1; j < L - 1; j++)
            for (int k = 1; k < L - 1; k++)
            {
                double tmp = fabs(B(i, j, k) - A(i, j, k));
                delta = Max(tmp, delta);
            }
    return delta;
}

void set(double *A, double *B) {
    for (int i = 0; i < L; i++)
        for (int j = 0; j < L; j++)
            for (int k = 0; k < L; k++)
            {
                A(i, j, k)= 0;
                if (i == 0 || j == 0 || k == 0 || i == L - 1 || j == L - 1 || k == L - 1)
                    B(i, j, k) = 0;
                else
                    B(i, j, k) = 4 + i + j + k;
            }    
}


class mabs{
public:
    __host__ __device__ float operator()(thrust::tuple<double,double> t) {
        float a, b;
        thrust::tie(a, b) = t;
        return fabs(a - b);
    }
};

int main(int an, char **as)
{
    std::cout << "usage:\t\tyacoby -[cg]" << std::endl;

    bool CPU = false;
    bool GPU = true;
    if (an >= 2) {
        GPU = false;
        for (int i = 0; as[1][i] != '\0'; ++i) {
            if (as[1][i] == 'g') GPU = true;
            if (as[1][i] == 'c') CPU = true;
        }
    }

    const long size = L * L * L * sizeof(double);
    
    double *A = (double*)malloc(size);
    double *B = (double*)malloc(size);

    float cpu_time = 0;
    if (CPU) {
        set(A, B);

        /* iteration loop */
        clock_t startt = clock();
        for (int it = 1; it <= ITMAX; it++) {
            eps = 0;
            
            for (int i = 1; i < L - 1; i++)
                for (int j = 1; j < L - 1; j++)
                    for (int k = 1; k < L - 1; k++)
                    {
                        double tmp = fabs(B(i, j, k) - A(i, j, k));
                        eps = Max(tmp, eps);
                        A(i, j, k) = B(i, j, k);
                    }

            for (int i = 1; i < L - 1; i++)
                for (int j = 1; j < L - 1; j++)
                    for (int k = 1; k < L - 1; k++)
                        B(i, j, k) = (A(i - 1, j, k) + A(i, j - 1, k) + A(i, j, k - 1) + A(i, j, k + 1) + A(i, j + 1, k) + A(i + 1, j, k)) / 6.0f;

            if (eps < MAXEPS)
                break;
        }
        clock_t endt = clock();

        cpu_time = float(endt - startt) / CLOCKS_PER_SEC;
    }

    double *A_host = (double*)malloc(size);
    double *B_host = (double*)malloc(size);

    float gpu_time = 0;
    if (GPU) {
        int deviceCount=0;
        hipGetDeviceCount( &deviceCount ); // число доступных устройств
        if (deviceCount < 1) exit(1);
        hipSetDevice(0); // Выбрать для работы заданное устройство
        
        set(A_host, B_host);

        double *A_device, *B_device;
        thrust::device_vector<double> A_thrust(L * L * L);
        A_device = (double*) thrust::raw_pointer_cast(&A_thrust[0]);
        thrust::device_vector<double> B_thrust(L * L * L);
        B_device = (double*) thrust::raw_pointer_cast(&B_thrust[0]);

        SAFE_CALL(hipMemcpy(A_device, A_host, size, hipMemcpyHostToDevice));
        SAFE_CALL(hipMemcpy(B_device, B_host, size, hipMemcpyHostToDevice));

        double eps = 0.0;


        dim3 blockDim = dim3(32, 4, 4);
        //int block = blockDim.x * blockDim.y * blockDim.z;
        dim3 gridDim = dim3(L / 32 + 1, L / 4 + 1, L / 4 + 1);

       
        bool flg = true;


        hipEvent_t startt, endt;
        hipEventCreate(&startt);
        hipEventCreate(&endt);

        hipEventRecord(startt, 0);

        flg = not flg;
        /* iteration loop */
        for (int it = 1; it <= ITMAX - 1; it++) {
            eps = thrust::transform_reduce(
                    thrust::make_zip_iterator(
                        thrust::make_tuple(
                            A_thrust.begin(),
                            B_thrust.begin()
                            )),
                    thrust::make_zip_iterator(
                        thrust::make_tuple(
                            A_thrust.end(),
                            B_thrust.end()
                            )),
                    mabs(),
                    0.0,
                    thrust::maximum<double>()
                    );
            if (flg)
                function<<<gridDim, blockDim>>>(A_device, B_device);
            else 
                function<<<gridDim, blockDim>>>(B_device, A_device);
            flg = not flg;
            if (eps < MAXEPS)
                break;
        }
        hipEventRecord(endt, 0);

        hipEventSynchronize(endt);
        hipEventElapsedTime(&gpu_time, startt, endt);
        hipEventDestroy(startt);
        hipEventDestroy(endt);

        if (flg)
            SAFE_CALL(hipMemcpy(A_host, A_device, size, hipMemcpyDeviceToHost));
        else
            SAFE_CALL(hipMemcpy(A_host, B_device, size, hipMemcpyDeviceToHost));

        SAFE_CALL(hipFree(A_device));
        SAFE_CALL(hipFree(B_device));
    }


    if (CPU && GPU) {
        std::cout << "cpu time = " << cpu_time << std::endl;
        std::cout << "gpu time = " << gpu_time * 0.001 <<  std::endl;
        std::cout << "decrease = " << cpu_time / gpu_time * 1000<< std::endl;
        std::cout << "maksimum deviation = " << dev(A, A_host) << std::endl;
    } else if (CPU) {
        std::cout << "cpu time = " << cpu_time << std::endl;
    } else if (GPU) {
        std::cout << "gpu time = " << gpu_time * 0.001 << std::endl;
    }

    free(A);
    free(A_host);
    free(B);
    free(B_host);

    return 0;
}
