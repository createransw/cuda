#include "hip/hip_runtime.h"
/* ADI program */

#include <cstdio>
#include <ctime>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/device_vector.h>

#define SAFE_CALL(err) do {                     \
    if (err != 0) {                             \
        std::cerr << "ERROR [" << __FILE__ << "] in line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl;         \
        exit(1);                                \
    }                                           \
} while(0)



#define Max(a, b) ((a) > (b) ? (a) : (b))

#define A(i, j, k) A[((i) * ny + (j)) * nx + (k)]
#define B(i, j, k) B[((i) * ny + (j)) * nx + (k)]
#define eps(i, j, k) eps[((i) * ny + (j)) * nx + (k)]

#define nx 31
#define ny 31
#define nz 31
        

double maxeps = 0.01;
double itmax = 2;

void init(double *a);
double dev(const double *A, const double *B);

__device__ int dim_i[nx / 32 + 1][ny / 32 + 1];
__device__ int dim_j[nx / 32 + 1][nz / 32 + 1];
__device__ int dim_k[ny / 32 + 1][nz / 32 + 1];

__global__ void function(double *A, double *eps, char dim) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;

    if (dim == 'i') {
        if ((threadIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0)) {
            while (atomicAdd(&dim_i[gridDim.x][gridDim.y], 0) < i);
        }
        __syncthreads();
        if ((threadIdx.x == 3) && (threadIdx.y == 17))
            printf("%d ", i);
        if ((i > 0) && (i < nx - 1))
            if ((j > 0) && (j < ny - 1))
                if ((k > 0) && (k < nz - 1))
                    A(i, j, k) = (A(i-1, j, k) + A(i+1, j, k)) / 2;
        __syncthreads();
        if ((threadIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0)) {
            __threadfence();
            atomicAdd(&dim_i[gridDim.x][gridDim.y], 1);
        }
    }

    if (dim == 'j') {
        if ((threadIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0)) {
            while (atomicAdd(&dim_j[gridDim.x][gridDim.z], 0) < j);
        }
        __syncthreads();
        if ((i > 0) && (i < nx - 1))
            if ((j > 0) && (j < ny - 1))
                if ((k > 0) && (k < nz - 1))
                    A(i, j, k) = (A(i, j-1, k) + A(i, j+1, k)) / 2; 
        __syncthreads();
        if ((threadIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0)) {
            __threadfence();
            atomicAdd(&dim_j[gridDim.x][gridDim.z], 1);
        }
    }

    if (dim == 'k') {
        if ((threadIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0)) {
            while (atomicAdd(&dim_k[gridDim.y][gridDim.z], 0) < k);
        }
        __syncthreads();
        if ((i > 0) && (i < nx - 1))
            if ((j > 0) && (j < ny - 1))
                if ((k > 0) && (k < nz - 1)) {
                    double tmp = (A(i, j, k-1) + A(i, j, k+1)) / 2;
                    eps(i, j, k) = fabs(A(i, j, k) - tmp);
                    A(i, j, k) = tmp;
                }
        __syncthreads();
        if ((threadIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0)) {
            __threadfence();
            atomicAdd(&dim_k[gridDim.y][gridDim.z], 1);
        }
    }
}


int main(int argc, char *argv[])
{
    std::cout << "usage:\t\tadi -[cg]" << std::endl;

    bool CPU = false;
    bool GPU = true;
    if (argc >= 2) {
        GPU = false;
        for (int i = 0; argv[1][i] != '\0'; ++i) {
            if (argv[1][i] == 'g') GPU = true;
            if (argv[1][i] == 'c') CPU = true;
        }
    }

    const long size = nx * ny * nz * sizeof(double);
    double *A = (double*)malloc(size);

    float cpu_time = 0;
    if (CPU) {
        init(A);

        clock_t startt = clock();
        for (int it = 1; it <= itmax; it++) {
            double eps = 0;        
            for (int i = 1; i < nx - 1; i++)
                for (int j = 1; j < ny - 1; j++)
                    for (int k = 1; k < nz - 1; k++)
                        A(i, j, k) = (A(i-1, j, k) + A(i+1, j, k)) / 2;

            for (int i = 1; i < nx - 1; i++)
                for (int j = 1; j < ny - 1; j++)
                    for (int k = 1; k < nz - 1; k++)
                        A(i, j, k) = (A(i, j-1, k) + A(i, j+1, k)) / 2; 

            for (int i = 1; i < nx - 1; i++)
                for (int j = 1; j < ny - 1; j++)
                    for (int k = 1; k < nz - 1; k++)
                    {
                        double tmp1 = (A(i, j, k-1) + A(i, j, k+1)) / 2;
                        double tmp2 = fabs(A(i, j, k) - tmp1);
                        eps = Max(eps, tmp2);
                        A(i, j, k) = tmp1;
                    }

            if (eps < maxeps)
                break;
        }
        clock_t endt = clock();

        cpu_time = float(endt - startt) / CLOCKS_PER_SEC;
    }

    double *A_host = (double*)malloc(size);

    float gpu_time = 0;
    if (GPU) {
        int deviceCount = 0;
        SAFE_CALL(hipGetDeviceCount(&deviceCount));
        if (deviceCount < 1) exit(1);
        SAFE_CALL(hipSetDevice(0));


        init(A_host);

        double *A_device;
        SAFE_CALL(hipMalloc((void**)&A_device, size));
        SAFE_CALL(hipMemcpy(A_device, A_host, size, hipMemcpyHostToDevice));


        thrust::device_vector<double> diff(nx * ny * nz);
        double *ptrdiff = thrust::raw_pointer_cast(&diff[0]);


        dim3 blockDim_k = dim3(1, 32, 32);
        dim3 gridDim_k = dim3(nx, ny / 32 + 1, nz / 32 + 1);

        dim3 blockDim_j = dim3(32, 1, 32);
        dim3 gridDim_j = dim3(nx / 32 + 1, ny, nz / 32 + 1);

        dim3 blockDim_i = dim3(32, 32, 1);
        dim3 gridDim_i = dim3(nx / 32 + 1, ny / 32 + 1, nz);

        int *dim_i_ptr; 
        SAFE_CALL(hipGetSymbolAddress((void**) &dim_i_ptr, dim_i));
        int *dim_j_ptr; 
        SAFE_CALL(hipGetSymbolAddress((void**) &dim_j_ptr, dim_j));
        int *dim_k_ptr; 
        SAFE_CALL(hipGetSymbolAddress((void**) &dim_k_ptr, dim_i));


        hipEvent_t startt, endt;
        SAFE_CALL(hipEventCreate(&startt));
        SAFE_CALL(hipEventCreate(&endt));

        double eps = 0;

        SAFE_CALL(hipEventRecord(startt, 0));
        for (int it = 1; it <= itmax; it++) {
            //std::cerr << "!";
            SAFE_CALL(hipMemset(dim_i_ptr, 0, sizeof(dim_i)));
            function<<<gridDim_i, blockDim_i>>>(A_device, ptrdiff, 'i');

            //std::cerr << "!";
            SAFE_CALL(hipMemset(dim_j_ptr, 0, sizeof(dim_j)));
            function<<<gridDim_j, blockDim_j>>>(A_device, ptrdiff, 'j');

            //std::cerr << "!";
            SAFE_CALL(hipMemset(dim_k_ptr, 0, sizeof(dim_k)));
            function<<<gridDim_k, blockDim_k>>>(A_device, ptrdiff, 'k');

            //std::cerr << "!";
            //std::cerr << it << ' ';

            eps = thrust::reduce(diff.begin(), diff.end(), 0.0, thrust::maximum<double>());
            if (eps < maxeps)
                break;
        }
        SAFE_CALL(hipEventRecord(endt, 0));

        SAFE_CALL(hipEventSynchronize(endt));
        SAFE_CALL(hipEventElapsedTime(&gpu_time, startt, endt));
        SAFE_CALL(hipEventDestroy(startt));
        SAFE_CALL(hipEventDestroy(endt));

        SAFE_CALL(hipMemcpy(A_host, A_device, size, hipMemcpyDeviceToHost));

        SAFE_CALL(hipFree(A_device));
    }

    if (CPU && GPU) {
        std::cout << "cpu time = " << cpu_time << std::endl;
        std::cout << "gpu time = " << gpu_time * 0.001 <<  std::endl;
        std::cout << "maksimum deviation = " << dev(A, A_host) << std::endl;
    } else if (CPU) {
        std::cout << "cpu time = " << cpu_time << std::endl;
    } else if (GPU) {
        std::cout << "gpu time = " << gpu_time * 0.001 << std::endl;
        std::cout << dev(A_host, A_host) << " zeros in matrix\n";
    }


    free(A);
    free(A_host);

    return 0;
}

void init(double *A)
{
    int i, j, k;
    for (i = 0; i < nx; i++)
        for (j = 0; j < ny; j++)
            for (k = 0; k < nz; k++)
                if (k == 0 || k == nz - 1 || j == 0 || j == ny - 1 || i == 0 || i == nx - 1)
                    A(i, j, k) = 10.0 * i / (nx - 1) + 10.0 * j / (ny - 1) + 10.0 * k / (nz - 1);
                else
                    A(i, j, k) = 0;
}

double dev(const double *A, const double *B) {
    double delta = 0.0;
    int count = 0;
    for (int i = 1; i < nx - 1; i++)
        for (int j = 1; j < ny - 1; j++)
            for (int k = 1; k < nz - 1; k++)
            {
                double tmp = fabs(B(i, j, k) - A(i, j, k));
                delta = Max(tmp, delta);

                if (A(i, j, k) == 0) count++;
            }
    return A == B ? count : delta;
}
