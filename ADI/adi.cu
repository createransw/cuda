#include "hip/hip_runtime.h"
/* ADI program */

#include <ctime>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/device_vector.h>

#define SAFE_CALL(err) do {                     \
    if (err != 0) {                             \
        std::cerr << "ERROR [" << __FILE__ << "] in line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl;         \
        exit(1);                                \
    }                                           \
} while(0)



#define Max(a, b) ((a) > (b) ? (a) : (b))

#define A(i, j, k) A[((i) * nx + (j)) * ny + (k)]
#define B(i, j, k) B[((i) * nx + (j)) * ny + (k)]
#define eps(i, j, k) eps[((i) * nx + (j)) * ny + (k)]

#define nx 384
#define ny 384
#define nz 384


double maxeps = 0.01;
double itmax = 100;

void init(double *a);
double dev(const double *A, const double *B);

__global__ void function(double *A, double *eps) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;

    if ((i >= 1) && (i < nx - 1))
        if ((j >= 1) && (j < ny - 1))
            if ((k >= 1) &&  (k < nz - 1))
                A(i, j, k) = (A(i-1, j, k) + A(i+1, j, k)) / 2;

    if ((i >= 1) && (i < nx - 1))
        if ((j >= 1) && (j < ny - 1))
            if ((k >= 1) && (k < nz - 1))
                A(i, j, k) = (A(i, j-1, k) + A(i, j+1, k)) / 2;

    if ((i >= 1) && (i < nx - 1))
        if ((j >= 1) && (j < ny - 1))
            if ((k >= 1) && (k < nz - 1))
            {
                double tmp1 = (A(i, j, k-1) + A(i, j, k+1)) / 2;
                eps(i, j, k) = fabs(A(i, j, k) - tmp1);
                A(i, j, k) = tmp1;
            }
}

int main(int argc, char *argv[])
{
    std::cout << "usage:\t\tadi -[cg]" << std::endl;

    bool CPU = false;
    bool GPU = true;
    if (argc >= 2) {
        GPU = false;
        for (int i = 0; argv[1][i] != '\0'; ++i) {
            if (argv[1][i] == 'g') GPU = true;
            if (argv[1][i] == 'c') CPU = true;
        }
    }

    const long size = nx * ny * nz * sizeof(double);
    double *A = (double*)malloc(size);

    float cpu_time = 0;
    if (CPU) {
        init(A);

        clock_t startt = clock();
        for (int it = 1; it <= itmax; it++) {
            double eps = 0;        
            for (int i = 1; i < nx - 1; i++)
                for (int j = 1; j < ny - 1; j++)
                    for (int k = 1; k < nz - 1; k++)
                        A(i, j, k) = (A(i-1, j, k) + A(i+1, j, k)) / 2;

            for (int i = 1; i < nx - 1; i++)
                for (int j = 1; j < ny - 1; j++)
                    for (int k = 1; k < nz - 1; k++)
                        A(i, j, k) = (A(i, j-1, k) + A(i, j+1, k)) / 2;

            for (int i = 1; i < nx - 1; i++)
                for (int j = 1; j < ny - 1; j++)
                    for (int k = 1; k < nz - 1; k++)
                    {
                        double tmp1 = (A(i, j, k-1) + A(i, j, k+1)) / 2;
                        double tmp2 = fabs(A(i, j, k) - tmp1);
                        eps = Max(eps, tmp2);
                        A(i, j, k) = tmp1;
                    }

            if (eps < maxeps)
                break;
        }
        clock_t endt = clock();

        cpu_time = float(endt - startt) / CLOCKS_PER_SEC;
    }

    double *A_host = (double*)malloc(size);

    float gpu_time = 0;
    if (GPU) {
        int deviceCount = 0;
        SAFE_CALL(hipGetDeviceCount(&deviceCount));
        if (deviceCount < 1) exit(1);
        SAFE_CALL(hipSetDevice(0));

        init(A_host);

        double *A_device;
        SAFE_CALL(hipMalloc((void**)&A_device, size));
        SAFE_CALL(hipMemcpy(A_device, A_host, size, hipMemcpyHostToDevice));

        thrust::device_vector<double> diff(nx * ny * nz);
        double *ptrdiff = thrust::raw_pointer_cast(&diff[0]);

        dim3 blockDim = dim3(8, 8, 8);
        dim3 gridDim = dim3(nx / 8 + 1, ny / 8 + 1, nz / 8 + 1);

        hipEvent_t startt, endt;
        SAFE_CALL(hipEventCreate(&startt));
        SAFE_CALL(hipEventCreate(&endt));

        SAFE_CALL(hipEventRecord(startt, 0));
        for (int it = 1; it <= itmax; it++) {
            function<<<gridDim, blockDim>>>(A_device, ptrdiff);
            double eps = thrust::reduce(diff.begin(), diff.end(), 0.0, thrust::maximum<double>());
            if (eps < maxeps)
                break;
        }
        SAFE_CALL(hipEventRecord(endt, 0));

        SAFE_CALL(hipEventSynchronize(endt));
        SAFE_CALL(hipEventElapsedTime(&gpu_time, startt, endt));
        SAFE_CALL(hipEventDestroy(startt));
        SAFE_CALL(hipEventDestroy(endt));

        SAFE_CALL(hipMemcpy(A_host, A_device, size, hipMemcpyDeviceToHost));

        SAFE_CALL(hipFree(A_device));
    }

    if (CPU && GPU) {
        std::cout << "cpu time = " << cpu_time << std::endl;
        std::cout << "gpu time = " << gpu_time * 0.001 <<  std::endl;
        std::cout << "maksimum deviation = " << dev(A, A_host) << std::endl;
    } else if (CPU) {
        std::cout << "cpu time = " << cpu_time << std::endl;
    } else if (GPU) {
        std::cout << "gpu time = " << gpu_time * 0.001 << std::endl;
    }


    free(A);
    free(A_host);

    return 0;
}

void init(double *A)
{
    int i, j, k;
    for (i = 0; i < nx; i++)
        for (j = 0; j < ny; j++)
            for (k = 0; k < nz; k++)
                if (k == 0 || k == nz - 1 || j == 0 || j == ny - 1 || i == 0 || i == nx - 1)
                    A(i, j, k) = 10.0 * i / (nx - 1) + 10.0 * j / (ny - 1) + 10.0 * k / (nz - 1);
                else
                    A(i, j, k) = 0;
}

double dev(const double *A, const double *B) {
    double delta = 0.0;
    for (int i = 1; i < nx - 1; i++)
        for (int j = 1; j < ny - 1; j++)
            for (int k = 1; k < nz - 1; k++)
            {
                double tmp = fabs(B(i, j, k) - A(i, j, k));
                delta = Max(tmp, delta);
            }
    return delta;
}

