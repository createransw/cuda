#include "hip/hip_runtime.h"
/* ADI program */

#include <cstdio>
#include <ctime>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/device_vector.h>

#define SAFE_CALL(err) do {                     \
    if (err != 0) {                             \
        std::cerr << "ERROR [" << __FILE__ << "] in line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl;         \
        exit(1);                                \
    }                                           \
} while(0)



#define Max(a, b) ((a) > (b) ? (a) : (b))

#define A(i, j, k) A[((i) * nx + (j)) * ny + (k)]
#define B(i, j, k) B[((i) * nx + (j)) * ny + (k)]
#define eps(i, j, k) eps[((i) * nx + (j)) * ny + (k)]

#define nx 100
#define ny 100
#define nz 100
        

double maxeps = 0.01;
double itmax = 10;

void init(double *a);
double dev(const double *A, const double *B);

__device__ int dim_count = 0;

__global__ void set() {
    dim_count = 0;
}

__global__ void function(double *A, double *eps, char dim) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;

    if (dim == 'i') {
        if ((threadIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0)) {
            while (atomicAdd(&dim_count, 0) < i * gridDim.x * gridDim.y);
        }
        __syncthreads();
        if ((i > 0) && (i < nx - 1))
            if ((j > 0) && (j < ny - 1))
                if ((k > 0) && (k < nz - 1))
                    A(i, j, k) = (A(i-1, j, k) + A(i+1, j, k)) / 2;
    }

    if (dim == 'j') {
        if ((threadIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0)) {
            while (atomicAdd(&dim_count, 0) < j * gridDim.x * gridDim.z);
        }
        __syncthreads();
        if ((i > 0) && (i < nx - 1))
            if ((j > 0) && (j < ny - 1))
                if ((k > 0) && (k < nz - 1))
                    A(i, j, k) = (A(i, j-1, k) + A(i, j+1, k)) / 2; 
    }

    if (dim == 'k') {
        if ((threadIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0)) {
            while (atomicAdd(&dim_count, 0) < k * gridDim.y * gridDim.z);
        }
        __syncthreads();
        if ((i > 0) && (i < nx - 1))
            if ((j > 0) && (j < ny - 1))
                if ((k > 0) && (k < nz - 1)) {
                    double tmp = (A(i, j, k-1) + A(i, j, k+1)) / 2;
                    eps(i, j, k) = fabs(A(i, j, k) - tmp);
                    A(i, j, k) = tmp;
                }
    }


    __syncthreads();
    if ((threadIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0)) {
        __threadfence();
        atomicAdd(&dim_count, 1);
    }
}


int main(int argc, char *argv[])
{
    std::cout << "usage:\t\tadi -[cg]" << std::endl;

    bool CPU = false;
    bool GPU = true;
    if (argc >= 2) {
        GPU = false;
        for (int i = 0; argv[1][i] != '\0'; ++i) {
            if (argv[1][i] == 'g') GPU = true;
            if (argv[1][i] == 'c') CPU = true;
        }
    }

    const long size = nx * ny * nz * sizeof(double);
    double *A = (double*)malloc(size);

    float cpu_time = 0;
    if (CPU) {
        init(A);

        clock_t startt = clock();
        for (int it = 1; it <= itmax; it++) {
            double eps = 0;        
            for (int i = 1; i < nx - 1; i++)
                for (int j = 1; j < ny - 1; j++)
                    for (int k = 1; k < nz - 1; k++)
                        A(i, j, k) = (A(i-1, j, k) + A(i+1, j, k)) / 2;

            for (int i = 1; i < nx - 1; i++)
                for (int j = 1; j < ny - 1; j++)
                    for (int k = 1; k < nz - 1; k++)
                        A(i, j, k) = (A(i, j-1, k) + A(i, j+1, k)) / 2; 

            for (int i = 1; i < nx - 1; i++)
                for (int j = 1; j < ny - 1; j++)
                    for (int k = 1; k < nz - 1; k++)
                    {
                        double tmp1 = (A(i, j, k-1) + A(i, j, k+1)) / 2;
                        double tmp2 = fabs(A(i, j, k) - tmp1);
                        eps = Max(eps, tmp2);
                        A(i, j, k) = tmp1;
                    }

            if (eps < maxeps)
                break;
        }
        clock_t endt = clock();

        cpu_time = float(endt - startt) / CLOCKS_PER_SEC;
    }

    double *A_host = (double*)malloc(size);

    float gpu_time = 0;
    if (GPU) {
        int deviceCount = 0;
        SAFE_CALL(hipGetDeviceCount(&deviceCount));
        if (deviceCount < 1) exit(1);
        SAFE_CALL(hipSetDevice(0));


        init(A_host);

        double *A_device;
        SAFE_CALL(hipMalloc((void**)&A_device, size));
        SAFE_CALL(hipMemcpy(A_device, A_host, size, hipMemcpyHostToDevice));


        thrust::device_vector<double> diff(nx * ny * nz);
        double *ptrdiff = thrust::raw_pointer_cast(&diff[0]);


        dim3 blockDim_k = dim3(1, 32, 32);
        dim3 gridDim_k = dim3(nx, ny / 32 + 1, nz / 32 + 1);

        dim3 blockDim_j = dim3(32, 1, 32);
        dim3 gridDim_j = dim3(nx / 32 + 1, ny, nz / 32 + 1);

        dim3 blockDim_i = dim3(32, 32, 1);
        dim3 gridDim_i = dim3(nx / 32 + 1, ny / 32 + 1, nz);


        hipEvent_t startt, endt;
        SAFE_CALL(hipEventCreate(&startt));
        SAFE_CALL(hipEventCreate(&endt));


        SAFE_CALL(hipEventRecord(startt, 0));
        for (int it = 1; it <= itmax; it++) {
            std::cerr << "!";
            set<<<1, 1>>>();
            function<<<gridDim_i, blockDim_i>>>(A_device, ptrdiff, 'i');
            std::cerr << "!";
            set<<<1, 1>>>();
            function<<<gridDim_j, blockDim_j>>>(A_device, ptrdiff, 'j');
            std::cerr << "!";
            set<<<1, 1>>>();
            function<<<gridDim_k, blockDim_k>>>(A_device, ptrdiff, 'k');
            std::cerr << "!";

            std::cerr << it << ' ';


            double eps = thrust::reduce(diff.begin(), diff.end(), 0.0, thrust::maximum<double>());
            if (eps < maxeps)
                break;
        }
        SAFE_CALL(hipEventRecord(endt, 0));

        SAFE_CALL(hipEventSynchronize(endt));
        SAFE_CALL(hipEventElapsedTime(&gpu_time, startt, endt));
        SAFE_CALL(hipEventDestroy(startt));
        SAFE_CALL(hipEventDestroy(endt));

        SAFE_CALL(hipMemcpy(A_host, A_device, size, hipMemcpyDeviceToHost));

        SAFE_CALL(hipFree(A_device));
    }

    if (CPU && GPU) {
        std::cout << "cpu time = " << cpu_time << std::endl;
        std::cout << "gpu time = " << gpu_time * 0.001 <<  std::endl;
        std::cout << "maksimum deviation = " << dev(A, A_host) << std::endl;
    } else if (CPU) {
        std::cout << "cpu time = " << cpu_time << std::endl;
    } else if (GPU) {
        std::cout << "gpu time = " << gpu_time * 0.001 << std::endl;
        std::cout << dev(A_host, A_host) << " zeros in matrix\n";
    }


    free(A);
    free(A_host);

    return 0;
}

void init(double *A)
{
    int i, j, k;
    for (i = 0; i < nx; i++)
        for (j = 0; j < ny; j++)
            for (k = 0; k < nz; k++)
                if (k == 0 || k == nz - 1 || j == 0 || j == ny - 1 || i == 0 || i == nx - 1)
                    A(i, j, k) = 10.0 * i / (nx - 1) + 10.0 * j / (ny - 1) + 10.0 * k / (nz - 1);
                else
                    A(i, j, k) = 0;
}

double dev(const double *A, const double *B) {
    double delta = 0.0;
    int count = 0;
    for (int i = 1; i < nx - 1; i++)
        for (int j = 1; j < ny - 1; j++)
            for (int k = 1; k < nz - 1; k++)
            {
                double tmp = fabs(B(i, j, k) - A(i, j, k));
                delta = Max(tmp, delta);

                if (A(i, j, k) == 0) count++;
            }
    return A == B ? count : delta;
}
